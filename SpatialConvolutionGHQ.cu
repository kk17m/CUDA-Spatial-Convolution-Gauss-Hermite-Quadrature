#include "hip/hip_runtime.h"
// *******************************************************
//   Spatial convolution using Gauss-Hermite quadrature
//
// *******************************************************

#include <iostream>
#include <math.h>
#include <fstream>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// CUDA thrust library
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

        //
        // Definig the CLOCK for performance testing.
        //
        long long wall_clock_time()
{
#ifdef __linux__
    struct timespec tp;
    clock_gettime(CLOCK_REALTIME, &tp);
    return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

//
// The parameters to compute the discrete points (Xn, Yn) are defined here.
// The axis limits along the x-axis are given by AXIS_MIN_X and AXIS_MAX_X, the
// axis limits along the y-axis are given by AXIS_MIN_Y and AXIS_MAX_Y.
//
// NOTE: These axis limits are not the limits of integration. The limits of
// integration are (-inf,inf).
//
// The number of discrete points Xn and Yn are given by NUM_PTS_X and NUM_PTS_Y.
// These points can have different sizes and should be a multiple of the
// BLOCK_SIZE in the respective dimension.
//
#define AXIS_MIN_X   -1
#define AXIS_MAX_X    1
#define AXIS_MIN_Y   -1
#define AXIS_MAX_Y    1
#define NUM_PTS_X 256
#define NUM_PTS_Y 256

//
// The CUDA parameters are defined here.
// The BLOCK_SIZE parameter for the CUDA x-dimension can be different than the
// CUDA y-dimension.
//
// The Z_BLOCK_SIZE should be a factor of sizeof(Gy)/sizeof(Gy[0]).
//
#define BLOCK_SIZE 16
#define Z_BLOCK_SIZE 4

//
// Define the Gauss-Hermite nodes n_i and weights w_i*exp((n_i)^2) for
// the two integrals. The size of Gy and Gx can be different depending on the
// required precision of the quadrature approximation.
//
__constant__ float Gy[36][2] = {{-7.626325754003896,0.8072646660353702},{-6.925598990259945,0.626482063538593},{-6.342243330994417,0.5482379054346118},{-5.818863279505579,0.5018952690514574},{-5.3335601071130645,0.4705108735743688},{-4.875039972467083,0.4476297837537447},{-4.436506970192858,0.4301720223313478},{-4.013456567749471,0.41645347099886904},{-3.6026938571484726,0.4054649988533432},{-3.201833945788157,0.3965612262672993},{-2.8090222351311054,0.38930924155705054},{-2.422766042053559,0.3834083398416976},{-2.0418271835544166,0.3786444980895176},{-1.6651500018434104,0.3748631855184457},{-1.2918109588209203,0.3719524810189278},{-0.9209818015707496,0.36983231208820944},{-0.5519014332904186,0.36844752436798417},{-0.18385336710581246,0.3677634858284455},{0.18385336710581512,0.36776348582843993},{0.5519014332904222,0.3684475243679883},{0.9209818015707576,0.3698323120882103},{1.2918109588209283,0.3719524810189504},{1.6651500018434149,0.3748631855184701},{2.0418271835544193,0.3786444980895354},{2.4227660420535626,0.38340833984170997},{2.8090222351311027,0.38930924155705887},{3.2018339457881595,0.3965612262673096},{3.6026938571484743,0.40546499885337384},{4.013456567749469,0.4164534709988875},{4.436506970192857,0.4301720223313582},{4.875039972467084,0.4476297837537804},{5.333560107113062,0.4705108735743989},{5.8188632795055755,0.5018952690514803},{6.342243330994413,0.5482379054346204},{6.925598990259943,0.6264820635385975},{7.6263257540038945,0.8072646660353738}};
__constant__ float Gx[36][2] = {{-7.626325754003896,0.8072646660353702},{-6.925598990259945,0.626482063538593},{-6.342243330994417,0.5482379054346118},{-5.818863279505579,0.5018952690514574},{-5.3335601071130645,0.4705108735743688},{-4.875039972467083,0.4476297837537447},{-4.436506970192858,0.4301720223313478},{-4.013456567749471,0.41645347099886904},{-3.6026938571484726,0.4054649988533432},{-3.201833945788157,0.3965612262672993},{-2.8090222351311054,0.38930924155705054},{-2.422766042053559,0.3834083398416976},{-2.0418271835544166,0.3786444980895176},{-1.6651500018434104,0.3748631855184457},{-1.2918109588209203,0.3719524810189278},{-0.9209818015707496,0.36983231208820944},{-0.5519014332904186,0.36844752436798417},{-0.18385336710581246,0.3677634858284455},{0.18385336710581512,0.36776348582843993},{0.5519014332904222,0.3684475243679883},{0.9209818015707576,0.3698323120882103},{1.2918109588209283,0.3719524810189504},{1.6651500018434149,0.3748631855184701},{2.0418271835544193,0.3786444980895354},{2.4227660420535626,0.38340833984170997},{2.8090222351311027,0.38930924155705887},{3.2018339457881595,0.3965612262673096},{3.6026938571484743,0.40546499885337384},{4.013456567749469,0.4164534709988875},{4.436506970192857,0.4301720223313582},{4.875039972467084,0.4476297837537804},{5.333560107113062,0.4705108735743989},{5.8188632795055755,0.5018952690514803},{6.342243330994413,0.5482379054346204},{6.925598990259943,0.6264820635385975},{7.6263257540038945,0.8072646660353738}};

//
// Declare the global vectors Xn, Yn, Cn here.
//
thrust::host_vector<float> Xn;
thrust::host_vector<float> Yn;
thrust::host_vector<float> Cn(NUM_PTS_X * NUM_PTS_Y);

//
// Define the function f(x,y) here.
//
__device__ float Fun(float x, float y)
{
    return exp(-(pow(x,2) + pow(y,2)));
}

//
// Define the convolution kernel g(x,y) here.
//
__device__ float Conv_Kernel(float x, float y)
{
    return exp(-(pow(x,2) + pow(y,2))/0.2);
}

//
// The inner quadrature sum, with weights wx and nodes nx, is computed here.
//
__device__ float Sum(float* ptrXn, float* ptrYn, float *ny, int *idx, int *idy)
{
    float nx, wx, Q1 = 0.0f;;

    int Nx = sizeof(Gx)/sizeof(Gx[0]);

    for (int k=0; k<Nx; k++)
    {
        nx = Gx[k][0];
        wx = Gx[k][1];
        Q1 +=  wx * Fun(nx, *ny) * Conv_Kernel(nx - ptrXn[*idx], *ny - ptrYn[*idy]) ;
    }

    return Q1;
}

//
// The CUDA kernel is defined here and the outer quadrature sum, with weights
// wy and nodes ny, is computed here.
//
__global__ void CUDA_kernel(float* ptrXn, float* ptrYn, float* ptrCn){

    int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int idz = blockIdx.z * Z_BLOCK_SIZE + threadIdx.z;

    float ny, wy;
    int stride_z = blockDim.z * gridDim.z;
    int Ny = sizeof(Gy)/sizeof(Gy[0]);

    while (idz < Ny ) {
        ny = Gy[idz][0];
        wy = Gy[idz][1];
        atomicAdd( &( ptrCn[idy * NUM_PTS_X + idx]), wy * Sum(ptrXn, ptrYn, &ny, &idx, &idy));
        idz += stride_z;
    }

}

int Kernelcall(){

    thrust::device_vector<float> d_Xn = Xn;
    thrust::device_vector<float> d_Yn = Yn;
    thrust::device_vector<float> d_Cn = Cn;

    float * ptrXn = thrust::raw_pointer_cast(&d_Xn[0]);
    float * ptrYn = thrust::raw_pointer_cast(&d_Yn[0]);
    float * ptrCn = thrust::raw_pointer_cast(&d_Cn[0]);

    int Ny = sizeof(Gy)/sizeof(Gy[0]);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, Z_BLOCK_SIZE);
    dim3 dimGrid((Xn.size() + dimBlock.x - 1) / dimBlock.x, (Yn.size() + dimBlock.y - 1) / dimBlock.y, (Ny + dimBlock.z - 1) / dimBlock.z);

    CUDA_kernel<<<dimGrid, dimBlock>>>(ptrXn, ptrYn, ptrCn);
    thrust::copy(d_Cn.begin(), d_Cn.end(), Cn.begin());

    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
        printf("Last CUDA error %s\n", hipGetErrorString(rc));

    //
    // Save result to a file
    //
    char buffer[32]; // The filename buffer.
    snprintf(buffer, sizeof(char) * 32, "FILE%i.txt", 0);
    std::ofstream out(buffer, std::ios_base::app);
    out.setf(std::ios::scientific);
    if( !out )
    {
        std::cout << "Couldn't open file."  << std::endl;
        return 1;
    }

    for (int i = 0; i < NUM_PTS_Y; i++) {
        for (int j = 0; j < NUM_PTS_X; j++) {
            out << Cn[i * NUM_PTS_X + j] <<',';
        }
        out <<'\n';
    }

    out.close();

    return 0;
}


//
// The main() function.
//
int main(int argc, char *argv[]){

    long long before, after;
    before = wall_clock_time();                                                                     // TIME START

    float xl = AXIS_MIN_X, xr = AXIS_MAX_X, yl = AXIS_MIN_Y, yr = AXIS_MAX_Y;
    int xpix = NUM_PTS_X, ypix = NUM_PTS_Y;

    thrust::host_vector<float> Del;
    Del.push_back((xr - xl) / xpix);
    Del.push_back((yr - yl) / ypix);

    for(int i=0; i < xpix; i++){
        Xn.push_back(xl + Del[0] * (i + 0.5));
    }

    for(int i=0; i < ypix; i++){
        Yn.push_back(yl + Del[1] * (i + 0.5));
    }

    Kernelcall();

    after = wall_clock_time();                                                                      // TIME END
    fprintf(stderr, "Process took %3.5f seconds ", ((float)(after - before))/1000000000);

    return 0;
}
